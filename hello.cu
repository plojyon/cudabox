// `nvcc hello.cu && ./a.out`
// https://www.nvidia.com/docs/io/116711/sc11-cuda-c-basics.pdf


#include <hip/hip_runtime.h>
#include <stdio.h>

// https://stackoverflow.com/a/14038590
#define gpuErrchk(ans) { gpuAssert((ans), __FILE__, __LINE__); }
inline void gpuAssert(hipError_t code, const char *file, int line, bool abort=true) {
	if (code != hipSuccess) {
		fprintf(stderr,"GPUassert: %s %s %d\n", hipGetErrorString(code), file, line);
		if (abort) exit(code);
	}
}


__global__ void add(int *a, int *b, int *c) {
	*c = *a + *b;
}

int main(void) {
	int a, b, c; // host copies of a, b, c
	int *d_a, *d_b, *d_c; // device copies of a, b, c
	int size = sizeof(int);
	// Allocate space for device copies of a, b, c
	hipMalloc((void **)&d_a, size);
	hipMalloc((void **)&d_b, size);
	hipMalloc((void **)&d_c, size);
	// Setup input values
	a = 2;
	b = 7;
	// Copy inputs to device
	hipMemcpy(d_a, &a, size, hipMemcpyHostToDevice);
	hipMemcpy(d_b, &b, size, hipMemcpyHostToDevice);
	// Launch add() kernel on GPU
	add<<<1,1>>>(d_a, d_b, d_c);
	// Copy result back to host
	hipMemcpy(&c, d_c, size, hipMemcpyDeviceToHost);
	// Cleanup
	hipFree(d_a); hipFree(d_b); hipFree(d_c);

	// Wait for the kernel to finish
	gpuErrchk(hipPeekAtLastError());
	gpuErrchk(hipDeviceSynchronize());

	printf("Result: %d + %d = %d\n", a, b, c);
	return 0;
}
